
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void kernel () {
  printf("Block/Thread: [%d, %d]\n", blockIdx.x, threadIdx.x);
}

__global__ void vectorAdd(const float *A, const float *B, float *C, int N) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < N) {
    C[i] = A[i] + B[i];
  }
}


int main() {
  hipError_t err = hipSuccess;
  int N = 50000;
  size_t size = N * sizeof(float);
  printf("[Vector addition of %d elements]\n", N);

  float *h_A = (float *)malloc(size);
  float *h_B = (float *)malloc(size);
  float *h_C = (float *)malloc(size);
  
  if (h_A == NULL || h_B == NULL || h_C == NULL) {
    fprintf(stderr, "Failed to allocate host vectors!\n");
    exit(EXIT_FAILURE);
  }

  for (int i = 0; i < N; ++i) {
    h_A[i] = rand() / (float)RAND_MAX;
    h_B[i] = rand() / (float)RAND_MAX;
  }

  float *d_A = NULL;
  err = hipMalloc((void **)&d_A, size);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  float *d_B = NULL;
  err = hipMalloc((void **)&d_B, size);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  float *d_C = NULL;
  err = hipMalloc((void **)&d_C, size);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }


  printf("Copy input data from the host memory to the CUDA device\n");
  err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Launch the Vector Add CUDA Kernel
  int threadsPerBlock = 256;
  int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
  printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
  vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, size);
  err = hipGetLastError();

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  
  printf("Copy output data from the CUDA device to the host memory\n");
  err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Verify that the result vector is correct
  for (int i = 0; i < N; ++i) {
    if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
      fprintf(stderr, "Result verification failed at element %d!\n", i);
      exit(EXIT_FAILURE);
    }
  }

  printf("Test PASSED\n");
  err = hipFree(d_A);
  err = hipFree(d_B);
  err = hipFree(d_C);
  free(h_A);
  free(h_B);
  free(h_C);

  printf("Done\n");
  return 0;
}
